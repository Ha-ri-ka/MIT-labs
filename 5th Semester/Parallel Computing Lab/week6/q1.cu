// Write a program in CUDA to count the number of times a given word is repeated in a sentence.
// (Use Atomic function)
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define N 1024
__global__ void CUDACount(char* A, unsigned int *d_count,char *sub,int len)
{
    int i = threadIdx.x;
    int flag;
    if(A[i]==sub[0])
    {
        flag=1;
        for(int j=1;j<len;j++)
        {
            if(A[i+j]!=sub[j])
            {
                flag=0;
                break;
            }
        }    
    }
    if(flag==1)
    atomicAdd(d_count,1);    
}
int main() 
{
    char A[N],sub[N];
    char *d_A,*d_sub;
    unsigned int *count=0,*d_count,*result;
    count=(unsigned int*)malloc(sizeof(unsigned int));
    result=(unsigned int*)malloc(sizeof(unsigned int));
    printf("enter the string: ");
    scanf("%[^\n]s",A);
    printf("enter substring: ");
    scanf("%[^\n]s",sub);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    hipMalloc((void**)&d_sub, strlen(sub)*sizeof(char));
    hipMalloc((void**)&d_A, strlen(A)*sizeof(char));
    hipMalloc((void **)&d_count,sizeof(unsigned int));
    hipMemcpy(d_sub, sub, strlen(sub)*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_A, A, strlen(A)*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count,count,sizeof(unsigned int),hipMemcpyHostToDevice);
    hipError_t error =hipGetLastError();
    if (error != hipSuccess) 
    {
        printf("CUDA Error1: %s\n", hipGetErrorString(error));
    }
    CUDACount<<<1,strlen(A)>>>(d_A,d_count,d_sub,strlen(sub));
    error =hipGetLastError();
    if (error != hipSuccess) 
    {
        printf("CUDA Error2: %s\n", hipGetErrorString(error));
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
    printf("Total occurences of %s=%u\n",sub,*result);
    printf("Time Taken=%f\n",elapsedTime);
    hipFree(d_A);
    hipFree(d_sub);
    hipFree(d_count);
    printf("\n");
    return 0;
}
