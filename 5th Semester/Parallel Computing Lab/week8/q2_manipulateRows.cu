//%%cuda --name rowman.cu
#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


__global__ void manipulateRows(int cols,int *mat, int*final)
{
    int m=blockDim.x;
    int row=blockIdx.x*blockDim.x+threadIdx.x;
    int i,k;
    if(row<m)
    {
        for(i=0;i<cols;i++)
        {
            k=row*m+i;
            final[k]=pow(mat[k],row+1);
        }
    }
}

int main()
{
    int m,n;
    printf("Enter number of rows:");
    scanf("%d",&m);
    printf("Enter number of columns:");
    scanf("%d",&n);
    //host variables
    int *mat=(int*)malloc(sizeof(int)*m*n);
    int *final=(int*)malloc(sizeof(int)*n);

    int i,j,k;
    printf("enter matrix row-wise:\n");
    for(i=0;i<n;i++)
    {
        for(j=0;j<n;j++)
        {
            k=i*n+j;
            scanf("%d",&mat[k]);
        }
    }
            
    //device variables
    int *d_mat,*d_final;
    hipMalloc((void**)&d_mat,sizeof(int)*m*n); 
    hipMalloc((void**)&d_final,sizeof(int)*m*n); 
    
    //copying values to device variables
    hipMemcpy(d_mat,mat,sizeof(int)*m*n,hipMemcpyHostToDevice); 


    //kernel Launch
    manipulateRows<<<1,m>>>(n,d_mat,d_final);

    hipMemcpy(final,d_final,sizeof(int)*m*n,hipMemcpyDeviceToHost);

    printf("\nResult:\n");
    for(i=0;i<m;i++)
    {
        for(j=0;j<n;j++)
        {
            k=i*m+j;
            printf("%d ",final[k]);
        }
        printf("\n");
    }
      
    hipFree(d_mat);
    hipFree(d_final);   
    return 0;
}
