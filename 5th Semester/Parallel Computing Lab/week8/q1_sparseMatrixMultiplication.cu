//%%cuda --name spvm.cu
#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"


__global__ void parallelSpVM(int n,int *data,int *col_ind,int *row_ptr,int *x,int *final)
{
    int ele_ind;
    int prod,row_start,row_end;
    int row=blockIdx.x*blockDim.x+threadIdx.x;
    if(row<n)
    {
        //printf("\n%d\n",row);
        prod=0;
        row_start=row_ptr[row];
        row_end=row_ptr[row+1];
        //printf("\nrow_start=%d,row_end=%d",row_start,row_end);
        for(ele_ind=row_start;ele_ind<row_end;ele_ind++)
        {
            //printf("\ndata=%d,x=%d,ele_ind=%d\n",data[ele_ind],x[ele_ind],ele_ind);
            prod+=data[ele_ind]*x[col_ind[ele_ind]];
        }
        //printf("\n%d\n",prod);
        final[row]=prod;
    }
}

int main()
{
    int n;
    printf("Enter the dimension of the sparse matrix (square):");
    scanf("%d",&n);
    //host variables
    int *sparse=(int*)malloc(sizeof(int)*n*n);
    int *data=(int*)malloc(sizeof(int)*n*n);
    int *col_ptr=(int*)malloc(sizeof(int)*n*n);
    int *row_ptr=(int*)malloc(sizeof(int)*(n+1));
    int *x=(int*)malloc(sizeof(int)*n);
    int *final=(int*)malloc(sizeof(int)*n);

    int i,j,k,l=0;
    int nonZeroCount=0;
    printf("enter sparse matrix:\n");
    for(i=0;i<n;i++)
    {
        row_ptr[l]=nonZeroCount;
        l++;
        for(j=0;j<n;j++)
        {
            k=i*n+j;
            scanf("%d",&sparse[k]);
            if(sparse[k]!=0)
            {
                data[nonZeroCount]=sparse[k];
                col_ptr[nonZeroCount]=j;
                nonZeroCount+=1;
            }
        }
    }
    row_ptr[n]=nonZeroCount;

    printf("\ndata array:\n");
    for (i = 0; i < nonZeroCount; i++)
        printf("%d ",data[i]);
    printf("\ncol_ind array:\n");
    for (i = 0; i < nonZeroCount; i++)
        printf("%d ",col_ptr[i]);
    printf("\nrow_ptr array:\n");
    for (i = 0; i < n+1; i++)
        printf("%d ",row_ptr[i]);

    printf("\nenter column vector:");
    for ( i = 0; i < n; i++)
        scanf("%d",&x[i]);
        
    //device variables
    int *d_data,*d_col_ptr,*d_row_ptr,*d_x,*d_final;
    hipMalloc((void**)&d_data,sizeof(int)*nonZeroCount);
    hipMalloc((void**)&d_col_ptr,sizeof(int)*nonZeroCount);
    hipMalloc((void**)&d_row_ptr,sizeof(int)*(n+1));
    hipMalloc((void**)&d_x,sizeof(int)*n);
    hipMalloc((void**)&d_final,sizeof(int)*n); 
  
    //copying values to device variables
    hipMemcpy(d_data,data,sizeof(int)*nonZeroCount,hipMemcpyHostToDevice); 
    hipMemcpy(d_col_ptr,col_ptr,sizeof(int)*nonZeroCount,hipMemcpyHostToDevice); 
    hipMemcpy(d_row_ptr,row_ptr,sizeof(int)*(n+1),hipMemcpyHostToDevice); 
    hipMemcpy(d_x,x,sizeof(int)*n,hipMemcpyHostToDevice); 

    //kernel Launch
    parallelSpVM<<<1,n>>>(n,d_data,d_col_ptr,d_row_ptr,d_x,d_final);

    hipMemcpy(final,d_final,sizeof(int)*n,hipMemcpyDeviceToHost);

    printf("\nResult:\n");
    for(int i = 0; i < n;i++)
      printf("%d ",final[i]);   
  
    hipFree(d_x);
    hipFree(d_final);
    hipFree(d_data);
    hipFree(d_row_ptr);
    hipFree(d_col_ptr);    
    return 0;
}
