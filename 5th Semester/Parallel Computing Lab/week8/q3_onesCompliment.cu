#include "hip/hip_runtime.h"
//%%cuda --name onescomp.cu
#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"
#include ""

__device__ int onesCompliment(int num)
{
    int onescomp = 0;
    int placeValue = 1;
    int r;
    while (num > 0) {
        int rem = num % 2;
        if(rem==1) r=0;
        else if(rem==0) r=1;
        onescomp = onecomp + r * placeValue;
        placeValue = placeValue * 10;
        num = num / 2;
    }
    return onescomp;
}

__global__ void manipulateValues(int cols,int *mat, int*final)
{
    int m=blockDim.x;
    int row=blockIdx.x*blockDim.x+threadIdx.x;
    int i,k;
    if(row<m)
    {
        if(row!=0 && row!=m-1) //border rows 
        {
            for(i=0;i<cols;i++)
            {
                k=row*m+i;
                if(i!=0 && i!=cols-1) //border columns                
                    final[k]=onesCompliment(mat[k]);                
                else
                    final[k]=mat[k];
            }            
        }
        else
        {
            for(i=0;i<cols;i++)
            {              
                k=row*m+i;
                final[k]=mat[k];                
            }     
        }
    }
}

int main()
{
    int m,n;
    printf("Enter number of rows:");
    scanf("%d",&m);
    printf("Enter number of columns:");
    scanf("%d",&n);
    //host variables
    int *mat=(int*)malloc(sizeof(int)*m*n);
    int *final=(int*)malloc(sizeof(int)*n);

    int i,j,k;
    printf("enter matrix row-wise:\n");
    for(i=0;i<n;i++)
    {
        for(j=0;j<n;j++)
        {
            k=i*n+j;
            scanf("%d",&mat[k]);
        }
    }
            
    //device variables
    int *d_mat,*d_final;
    hipMalloc((void**)&d_mat,sizeof(int)*m*n); 
    hipMalloc((void**)&d_final,sizeof(int)*m*n); 
    
    //copying values to device variables
    hipMemcpy(d_mat,mat,sizeof(int)*m*n,hipMemcpyHostToDevice); 


    //kernel Launch
    manipulateValues<<<1,m>>>(n,d_mat,d_final);

    hipMemcpy(final,d_final,sizeof(int)*m*n,hipMemcpyDeviceToHost);

    printf("\nResult:\n");
    for(i=0;i<m;i++)
    {
        for(j=0;j<n;j++)
        {
            k=i*m+j;
            printf("%d ",final[k]);
        }
        printf("\n");
    }
      
    hipFree(d_mat);
    hipFree(d_final);   
    return 0;
}
