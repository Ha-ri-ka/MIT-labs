// Implement a CUDA program to add two vectors of length N by keeping the number of
// threads per block as 256 (constant) and vary the number of blocks to handle N elements.
#include<stdio.h>	
#include<math.h>
#include "hip/hip_runtime.h"

__global__ void add(int *a, int *b, int *c)
{    
    int tid=(blockIdx.x*blockDim.x)+threadIdx.x;
    //printf("blockid:%d,threadid:%d\n",blockIdx.x,threadIdx.x);
    c[tid]=a[tid]+b[tid];
}		
int main(void) 
{
    int n;
    printf("Enter size of array:"); 
    scanf("%d",&n);
    int size = n * sizeof(int);
    int a[n], b[n], c[n], i;
    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    printf("\nEnter matrix A:\n");
    for(i=0;i<n;i++)
    {
        printf("a[%d]=",i);
        scanf("%d",&a[i]);
    }
    printf("Enter matrix B:\n");
    for(i=0;i<n;i++)
    {
        printf("b[%d]=",i);
        scanf("%d",&b[i]);
    }
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
    //efficient dimensions
    dim3 eff_dimGrid(ceil(n/256.0),1,1);
    dim3 eff_dimBlock(256,1,1);
    //1 block n threads
    dim3 a_dimGrid(1,1,1);
    dim3 a_dimBlock(n,1,1);
    //n blocks 1 thread in each
    dim3 b_dimGrid(n,1,1);
    dim3 b_dimBlock(1,1,1);
    printf("\nEfficient thread and block creation.\n");
    add<<<eff_dimGrid,eff_dimBlock>>>(d_a, d_b, d_c);
    printf("Result:\n");
    for(i=0;i<n;i++)
    printf("%d ",c[i]);
    printf("\n1 block with n threads.\n");
    add<<<a_dimGrid,a_dimBlock>>>(d_a, d_b, d_c);
    printf("Result:\n");
    for(i=0;i<n;i++)
    printf("%d ",c[i]);
    printf("\nN block with 1 thread in each.\n");
    add<<<b_dimGrid,b_dimBlock>>>(d_a, d_b, d_c);	
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    printf("Result:\n");
    for(i=0;i<n;i++)
    printf("%d ",c[i]);    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}