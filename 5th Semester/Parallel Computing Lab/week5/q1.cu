// Write a program in CUDA to add two vectors of length N using
// a) block size as N
// b) N threads
#include<stdio.h>	
#include "hip/hip_runtime.h"

__global__ void add(int *a, int *b, int *c)
{    
    int tid=(blockIdx.x*blockDim.x)+threadIdx.x;
    printf("blockid:%d,threadid:%d\n",blockIdx.x,threadIdx.x);
    c[tid]=a[tid]+b[tid];
}		
int main(void) 
{
    int n;
    printf("Enter size of array:"); 
    scanf("%d",&n);
    int size = n * sizeof(int);
    int a[n], b[n], c[n], i;
    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    printf("\nEnter matrix A:\n");
    for(i=0;i<n;i++)
    {
        printf("a[%d]=",i);
        scanf("%d",&a[i]);
    }
    printf("\nEnter matrix B:\n");
    for(i=0;i<n;i++)
    {
        printf("b[%d]=",i);
        scanf("%d",&b[i]);
    }
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
    //1 block
    dim3 dimGrid(1,1,1);
    //n blocks
    //dim3 dimGrid(n,1,1);
    //n threads per block
    dim3 dimBlock(n,1,1);
    //1 thread per block
    //dim3 dimBlock(1,1,1);
    add<<<dimGrid,dimBlock>>>(d_a, d_b, d_c);	
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    printf("Result:\n");
    for(i=0;i<n;i++)
    printf("%d ",c[i]);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}