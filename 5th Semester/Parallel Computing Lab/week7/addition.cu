#include "hip/hip_runtime.h"
//%%cuda --name addition.cu
#include <stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
__global__ void addRow(int *mat1,int *mat2, int *final, int cols)
{
    int tid=blockIdx.x*blockDim.x+threadIdx.x;
    int row;
    if(tid!=0)
      row=tid+cols;
    else if(tid==0)
      row=0;
    int i,k;
    for(i=0;i<cols;i++)
    {
        k=row+i;
        final[k]=mat1[k]+mat2[k];
    }
}
__global__ void addCol(int *mat1,int *mat2, int *final, int rows)
{
    int tid=blockIdx.x*blockDim.x+threadIdx.x;
    int cols=blockDim.x;
    int i,k;
    final[tid]=mat1[tid]+mat2[tid];
    for(i=0;i<rows-1;i++)
    {
        k=tid+cols;
        final[k]=mat1[k]+mat2[k];
    }
}
__global__ void addEles(int *mat1,int *mat2,int *final)
{
    int tid=blockIdx.x*blockDim.x+threadIdx.x;
    final[tid]=mat1[tid]+mat2[tid];
}
int main()
{
    int r,c,i,j,k;
    printf("number of rows: "); 
    scanf("%d",&r);
    printf("enter number of columns: ");
    scanf("%d",&c);
    int * mat1=(int*)malloc(sizeof(int)*r*c);
    int * mat2=(int*)malloc(sizeof(int)*r*c);
    int * final=(int*)malloc(sizeof(int)*r*c);
    printf("enter matrix 1:\n");
    for(i=0;i<r;i++)
    {
        for(j=0;j<c;j++)
        {
            k = i * r + j;
            scanf("%d", &mat1[k]);
        }
    }
    printf("enter matrix 2:\n");
    for(i=0;i<r;i++)
    {
        for(j=0;j<c;j++)
        {
            k = i * r + j;
            scanf("%d", &mat2[k]);
        }
    }

    int * d_mat1,*d_mat2,*d_final;
    hipMalloc((void**)&d_mat1,sizeof(int)*r*c);
    hipMalloc((void**)&d_mat2,sizeof(int)*r*c);
    hipMalloc((void**)&d_final,sizeof(int)*r*c);

    hipMemcpy(d_mat1,mat1,sizeof(int)*r*c,hipMemcpyHostToDevice);
    hipMemcpy(d_mat2,mat2,sizeof(int)*r*c,hipMemcpyHostToDevice);

    printf("Each thread calculates one row\n")
    addRow<<<1,r>>>(d_mat1,d_mat2,d_final,c);
    hipMemcpy(final,d_final,sizeof(int)*r*c,hipMemcpyDeviceToHost);
    printf("resultant matrix:\n");
    for(i=0;i<r;i++)
    {
        for(j=0;j<c;j++)
        printf("%d ",final[i]);
        printf("\n");
    }

    printf("Each thread calculates one column\n");
    addCol<<<1,c>>>(d_mat1,d_mat2,d_final,r);
    hipMemcpy(final,d_final,sizeof(int)*r*c,hipMemcpyDeviceToHost);
    printf("resultant matrix:\n");
    for(i=0;i<r;i++)
    {
        for(j=0;j<c;j++)
        printf("%d ",final[i]);
        printf("\n");
    }

    printf("Each thread calculates one Element\n");
    addEles<<<1,r*c>>>(d_mat1,d_mat2,d_final);
    hipMemcpy(final,d_final,sizeof(int)*r*c,hipMemcpyDeviceToHost);
    printf("resultant matrix:\n");
    for(i=0;i<r;i++)
    {
        for(j=0;j<c;j++)
        printf("%d ",final[i]);
        printf("\n");
    }

    hipFree(d_mat1);
    hipFree(d_mat2);
    hipFree(d_final);
    return 0;
}
